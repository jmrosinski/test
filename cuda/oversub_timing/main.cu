#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <gptl.h>
#include <gptl_cuda.h>

__global__ void set_baseclock (long long *);
__global__ void sub (long long *, long long *, long long *, int, int, float);
__device__ long long *baseclock;

// managed variables
static long long *minstamp;
static long long *maxstamp;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
  {
    if (code != hipSuccess) 
      {
	fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
      }
  }

int main ()
{
  const int smcount      = 5;
  const int warps_per_sm = 4;
  const int warpsize     = 32;
  int w, sm, i;
  int oversub;
  int nblocks;
  hipError_t cudaret;
  int ret;
  float sleepsec;

  printf ("Enter oversubscription factor\n");
  scanf ("%d", &oversub);
  printf ("oversub=%d\n", oversub);

  printf ("Enter sleep time in floating point seconds\n");
  scanf ("%f", &sleepsec);
  printf ("sleepsec=%f\n", sleepsec);

  // Need GPTL lib for GPTLmy_sleep
  ret = GPTLinitialize ();

  gpuErrchk (hipMalloc (&baseclock, smcount * sizeof (long long)));

  nblocks = oversub * smcount * warps_per_sm;
  gpuErrchk (hipMallocManaged (&minstamp, nblocks * sizeof (long long)));
  gpuErrchk (hipMallocManaged (&maxstamp, nblocks * sizeof (long long)));
  for (i = 0; i < nblocks; ++i) {
    minstamp[i] = LLONG_MAX;
    maxstamp[i] = 0;
  }
  hipDeviceSynchronize ();

  // Set baseclock for each SM
  set_baseclock <<<smcount,warpsize*warps_per_sm>>> (baseclock);
  cudaret = hipGetLastError();
  printf("%s\n", hipGetErrorString(cudaret));
  hipDeviceSynchronize ();
  cudaret = hipGetLastError();
  printf("%s\n", hipGetErrorString(cudaret));

  sub <<<nblocks,warpsize>>> (baseclock, minstamp, maxstamp, warps_per_sm, warpsize, sleepsec);
  cudaret = hipGetLastError();
  printf("%s\n", hipGetErrorString(cudaret));
  hipDeviceSynchronize ();
  cudaret = hipGetLastError();
  printf("%s\n", hipGetErrorString(cudaret));

  hipDeviceSynchronize ();

  for (sm = 0; sm < smcount; ++sm) {
    for (w = 0; w < warps_per_sm; ++w) {
      int idx = sm*warps_per_sm + w;
      if (maxstamp[idx] > 0) {
	printf ("w=%d sm=%d minstamp=%lld maxstamp=%lld\n", w, sm, minstamp[idx], maxstamp[idx]);
      }
    }
  }
}

__global__ void set_baseclock (long long *baseclock)
{
  uint smid;

  if (threadIdx.x > 0)
    return;
  if (blockDim.y != 1 || blockDim.z != 1) {
    printf ("blockDim.y and blockDim.z must all be 1\n");
    return;
  }

  asm volatile ("mov.u32 %0, %smid;" : "=r"(smid));
  baseclock[smid] = clock64();
}

__global__ void sub (long long *baseclock, long long *minstamp, long long *maxstamp,
		     int warps_per_sm, int warpsize, float sleepsec)
{
  uint smid;        // 0-smcount
  long long stamp;  // time stamp
  int idx;          // index into 2d array
  int w = (threadIdx.x + blockIdx.x*blockDim.x) / warpsize;  // linearized warp number

  if (threadIdx.x > 0)
    return;
  if (blockDim.y != 1 || blockDim.z != 1) {
    printf ("blockDim.y and blockDim.z must all be 1\n");
    return;
  }

  asm volatile ("mov.u32 %0, %smid;" : "=r"(smid));
  printf ("sub: smid=%d w=%d\n", smid, w);
  
  idx = smid*warps_per_sm + w;
  
  stamp = clock64() - baseclock[smid];
  if (stamp < minstamp[idx])
    minstamp[idx] = stamp;

  GPTLmy_sleep (sleepsec);

  stamp = clock64() - baseclock[smid];
  if (stamp > maxstamp[idx])
    maxstamp[idx] = stamp;

  __syncthreads ();
}
