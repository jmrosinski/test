#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void Kernel1 (int nz, int nip, int *d_kprox, float *d_arr)
{
  int ipn, ik, isn, k;

  ipn = blockIdx.x;
  k = threadIdx.x;

  ik = ipn*nz + k;
  d_arr[ik] = 0.;
  for (isn = 0; isn < d_kprox[k]; isn++) {
    d_arr[ik] += 1.;
  }
}

__host__ int main ()
{
  const int nz = 192;    // number of vertical points (threading)
  const int nip = 10242; // number of horizontal points (blocking)

  int k;                 // vertical index
  int ik;                // loop index collapses ipn and k
  int ipn;               // loop index over nip
  int isn;               // loop index over kprox
  int kprox[nz];         // mimics kprox in NIM
  int *d_kprox;          // kprox on device

  float refarr[nip*nz];  // reference array on CPU containing correct results
  float outarr[nip*nz];  // output array from GPU containing test results
  float *d_arr;

  dim3 cuda_threads1(nz);   // nz threads
  dim3 cuda_grids1(nip);    // nip blocks

  for (k = 0; k < nz; ++k) {
    kprox[k] = (k%6) + 1;
    for (ipn = 0; ipn < nip; ++ipn) {
      ik = ipn*nz + k;
      refarr[ik] = 0.;
      for (isn = 0; isn < kprox[k]; isn++) {
	refarr[ik] += 1.;
      }
    }
  }

  // Allocate space on device
  hipMalloc ((void **) &d_arr,   nz*nip*sizeof(float));
  hipMalloc ((void **) &d_kprox, nz*sizeof(int));

  // Set up kprox array for device
  hipMemcpy (d_kprox, kprox, nz*sizeof(int), hipMemcpyHostToDevice);

  Kernel1<<< cuda_grids1, cuda_threads1 >>>(nz, nip, d_kprox, d_arr);
  hipDeviceSynchronize ();

  // Copy results back to host
  hipMemcpy (outarr, d_arr, nz*nip*sizeof(float), hipMemcpyDeviceToHost);

  // Check results
  for (k = 0; k < nz; ++k) {
    for (ipn = 0; ipn < nip; ++ipn) {
      ik = ipn*nz + k;
      if (outarr[ik] != refarr[ik]) {
	printf ("ik=%d outarr=%f refarr=%f\n", ik, outarr[ik], refarr[ik]);
	return -1;
      }
    }
  }
  return 0;
}
