#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void Kernel1 (int nz, int nip, int *d_nprox, float *d_arr)
{
  int ipn, ik, isn, k;

  ipn = blockIdx.x;
  k = threadIdx.x;

  //  if (k == 0)
  //    printf ("Kernel1: nprox[%d]=%d\n", ipn, d_nprox[ipn]);

  ik = ipn*nz + k;
  d_arr[ik] = 0.;
  for (isn = 0; isn < d_nprox[ipn]; isn++) {
    d_arr[ik] += 1.;
  }
  //  printf ("Kernel1: k=%d ik=%d d_arr=%f\n", k, ik, d_arr[ik]);
}

__host__ int main ()
{
  const int nz = 192;    // number of vertical points (threading)
  const int nip = 10242; // number of horizontal points (blocking)

  int k;                 // vertical index
  int ik;                // loop index collapses ipn and k
  int ipn;               // loop index over nip
  int isn;               // loop index over nprox
  int nprox[nip];        // mimics nprox in NIM
  int *d_nprox;          // nprox on device

  float refarr[nip*nz];  // reference array on CPU containing correct results
  float outarr[nip*nz];  // output array from GPU containing test results
  float *d_arr;

  dim3 cuda_threads1(nz);   // nz threads
  dim3 cuda_grids1(nip);    // nip blocks

  for (ipn = 0; ipn < nip; ++ipn) {
    nprox[ipn] = (ipn%6) + 1;
    for (k = 0; k < nz; ++k) {
      ik = ipn*nz + k;
      refarr[ik] = 0.;
      for (isn = 0; isn < nprox[ipn]; isn++) {
	refarr[ik] += 1.;
      }
    }
  }

  // Allocate space on device
  hipMalloc ((void **) &d_arr,   nz*nip*sizeof(float));
  hipMalloc ((void **) &d_nprox, nip*sizeof(int));

  // Set up nprox array for device
  hipMemcpy (d_nprox, nprox, nip*sizeof(int), hipMemcpyHostToDevice);

  Kernel1<<< cuda_grids1, cuda_threads1 >>>(nz, nip, d_nprox, d_arr);
  hipDeviceSynchronize ();

  // Copy results back to host
  hipMemcpy (outarr, d_arr, nz*nip*sizeof(float), hipMemcpyDeviceToHost);

  // Check results
  for (ipn = 0; ipn < nip; ++ipn) {
    for (k = 0; k < nz; ++k) {
      ik = ipn*nz + k;
      if (outarr[ik] != refarr[ik]) {
	printf ("ik=%d outarr=%f refarr=%f\n", ik, outarr[ik], refarr[ik]);
	return -1;
      }
    }
  }
  return 0;
}
