
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ static int *arrptr;
__device__ static int x;

extern "C" {
__device__ void sub1_()
{
  arrptr = (int *) malloc (10);
  x = 11;
  printf ("sub1: arrptr=%p\n", arrptr);
  printf ("sub1: x=%d\n", x);
}


__device__ void sub2_()
{
  printf ("sub2: arrptr=%p\n", arrptr);
  printf ("sub2: x=%d\n", x);
}
}
