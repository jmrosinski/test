#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gptl.h>
#include <gptl_cuda.h>

__global__ void runit (float, float, int *, int *);
__global__ void setup_handles (int *, int *);

int *runit_handle;
int *percall_handle;

int main ()
{
  int blocksize;
  int warpsize = -1;
  int khz = -1;
  int devnum = -1;
  int smcount = -1;
  int cores_per_sm = -1;
  int cores_per_gpu = -1;
  int oversub = -1;
  int nwarps;
  int nthreads;
  int nblocks;
  
  int ret;
  float sleep_tot;
  float sleep_percall;

  ret = GPTLget_gpu_props (&khz, &warpsize, &devnum, &smcount, &cores_per_sm, &cores_per_gpu);

  printf ("Enter oversubsubscription factor\n");
  scanf ("%d", &oversub);
  printf ("oversub=%d\n", oversub);

  nwarps = (cores_per_gpu * oversub) / warpsize;
  printf ("nwarps=%d\n", nwarps);
  if (nwarps * warpsize != cores_per_gpu * oversub)
    printf ("NOTE: warpsize=%d does not divide evenly into cores_per_gpu(%d) * oversub(%d)=%d\n",
	    warpsize, cores_per_gpu, oversub, cores_per_gpu * oversub);
  ret = GPTLsetoption (GPTLmaxwarps_gpu, nwarps);

  printf ("Enter sleep_tot sleep_percall (both in floating point seconds)\n");
  scanf ("%f%f", &sleep_tot, &sleep_percall);
  printf ("sleep_tot=%f sec sleep_percall=%f sec\n", sleep_tot, sleep_percall);

  ret = GPTLinitialize ();

  (void) (hipMallocManaged (&runit_handle,   sizeof (int)));
  (void) (hipMallocManaged (&percall_handle, sizeof (int)));

  setup_handles <<<1,1>>> (runit_handle, percall_handle);
  hipDeviceSynchronize ();

  blocksize = cores_per_sm;
  nthreads = nwarps * warpsize;
  nblocks = nthreads / blocksize;

  printf ("nblocks=%d blocksize=%d\n", nblocks, blocksize);
  ret = GPTLstart ("total");
  runit <<<nblocks,blocksize>>> (sleep_tot, sleep_percall, runit_handle, percall_handle);
  hipDeviceSynchronize ();
  ret = GPTLstop ("total");
  ret = GPTLpr (0);
  return 0;
}

__global__ void setup_handles (int *runit_handle, int *percall_handle)
{
  int ret;
  
  ret = GPTLinit_handle_gpu ("runit",   runit_handle);
  ret = GPTLinit_handle_gpu ("percall", percall_handle);
}

__global__ void runit (float sleep_tot, float sleep_percall, int *runit_handle, int *percall_handle)
{
  int ret;
  double slept = 0.;

  ret = GPTLstart_gpu (*runit_handle);
  while (slept < sleep_tot) {
    ret = GPTLstart_gpu (*percall_handle);
    ret = GPTLmy_sleep (sleep_percall);
    ret = GPTLstop_gpu (*percall_handle);
    slept += sleep_percall;
  }
  ret = GPTLstop_gpu (*runit_handle);
}
