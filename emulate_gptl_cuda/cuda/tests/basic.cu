#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../src/gptl_cuda.h"

__global__ void runit (void);

int main ()
{
  int ret;
  int nblocks   = 2;
  int blocksize = 3;

  runit<<<nblocks,blocksize>>> ();
  hipDeviceSynchronize ();
  return 0;
}

__global__ void runit (void)
{
  int ret;
  
  printf ("Entered global routine runit\n");
  ret = GTLstart_gpu (0);
  ret = GTLstop_gpu (1);
}
